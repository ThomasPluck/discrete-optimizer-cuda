#include <stdio.h>
#include <string>
#include <iostream>
#include <vector>

#include "util.cuh"
#include "layer.h"

#include "data_utils.h"

using namespace std;



int main()
{    
    int dev = 0;
    hipSetDevice(dev);
    const unsigned batch = 32;
    const unsigned image_height = 28;
    const unsigned image_width = 28;
    const unsigned data_length = 60000;

    const unsigned data_threshold = 50;

    // =============== Get Input and Label =================
    uchar ** ar = ReadMNIST(".data/mnist/train-images.idx3-ubyte");

    // ================= Set Network =================

    CUDA_SAFE_CALL(hipGetDeviceProperties(&Launch::deviceProp, 0));

    FcLayer layer1 = FcLayer(image_height*image_width,10,batch);

    layer1.weights.fill_random();
    layer1.biases.fill(128);
    layer1.weight_counters.fill();
    layer1.bias_counters.fill();

    uchar ** batch_slice;
    batch_slice = new uchar*[batch];
    for (int i = 0; i < image_height * image_width; i++) {
        batch_slice[i] = new uchar[image_height * image_width];
    }

    // ================= Train Network =================

    std::cout << "Training Network..." << std::endl;
    for(int i = 0; i < data_length/batch; i++){

        // Get batched data into 2D array
        for(int j = 0; j < batch*image_height*image_width; j++) {
            int row = j % (image_height*image_width);
            int col = j / (image_height*image_width);
            batch_slice[row][col] = ar[row+i*batch][col];
        }

        Host_Matrix train_batch = ThresholdAndPack(batch_slice,data_threshold);

        layer1.input = train_batch;
        layer1.input.upload();

        layer1.forward();
        layer1.back();
    }
    
    return 0;
}